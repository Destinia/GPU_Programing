#include "hip/hip_runtime.h"
#include "lab2.h"
#include <cmath>
#include <random>
#include <algorithm>
#include <vector>
#include <fstream>
#include <iostream>
#include <numeric>
#include "../utils/SyncedMemory.h"
static const unsigned W = 640;
static const unsigned H = 480;
static const unsigned NFRAME = 240;

using namespace std;

//////////////initial////


struct Lab2VideoGenerator::Impl {
	int t = 0;
	vector<int> p;
};

__device__ double fade(double t) { 
	return t * t * t * (t * (t * 6 - 15) + 10);
}

__device__ double lerp(double t, double a, double b) { 
	return a + t * (b - a); 
}

__device__ double grad(int hash, double x, double y, double z) {
	int h = hash & 15;
	// Convert lower 4 bits of hash inot 12 gradient directions
	double u = h < 8 ? x : y,
		   v = h < 4 ? y : h == 12 || h == 14 ? x : z;
	return ((h & 1) == 0 ? u : -u) + ((h & 2) == 0 ? v : -v);
}

__global__ void noise(double z,const int* p ,uint8_t* yuv) {
	//printf("in\n");
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	double x = (double)(idx%W)/W;
	double y = (double)(idx/W)/H;
	// Find the unit cube that contains the point
	int X = (int) floor(x) & 255;
	int Y = (int) floor(y) & 255;
	int Z = (int) floor(z) & 255;

	// Find relative x, y,z of point in cube
	x -= floor(x);
	y -= floor(y);
	z -= floor(z);

	// Compute fade curves for each of x, y, z
	double u = fade(x);
	double v = fade(y);
	double w = fade(z);

	// Hash coordinates of the 8 cube corners
	int A = p[X] + Y;
	int AA = p[A] + Z;
	int AB = p[A + 1] + Z;
	int B = p[X + 1] + Y;
	int BA = p[B] + Z;
	int BB = p[B + 1] + Z;

	// Add blended results from 8 corners of cube
	double res = lerp(w, lerp(v, lerp(u, grad(p[AA], x, y, z), grad(p[BA], x-1, y, z)), lerp(u, grad(p[AB], x, y-1, z), grad(p[BB], x-1, y-1, z))),	lerp(v, lerp(u, grad(p[AA+1], x, y, z-1), grad(p[BA+1], x-1, y, z-1)), lerp(u, grad(p[AB+1], x, y-1, z-1),	grad(p[BB+1], x-1, y-1, z-1))));
	res = (res + 1.0)/2.0;
	//origin
	//yuv[idx] = floor((res + 1.0)/2.0*255);
	//yuv[idx] = (res + 1.0)/2.0;
	//typical noise
	//wood like
	double n = (60+30*z) * res;
	n = n - floor(n);

	yuv[idx] = uint8_t(floor(150 * n));
			
}


Lab2VideoGenerator::Lab2VideoGenerator(): impl(new Impl) {
	unsigned int seed = 237;
	
	impl->p.resize(256);

	// Fill p with values from 0 to 255
	std::iota(impl->p.begin(), impl->p.end(), 0);

	// Initialize a random engine with seed
	std::default_random_engine engine(seed);

	// Suffle  using the above random engine
	std::shuffle(impl->p.begin(), impl->p.end(), engine);

	// Duplicate the permutation vector
	impl->p.insert(impl->p.end(), impl->p.begin(), impl->p.end());
	
	printf("output test\n");

	printf("Initialize finish\n" );
	
}

Lab2VideoGenerator::~Lab2VideoGenerator() {

}

void Lab2VideoGenerator::get_info(Lab2VideoInfo &info) {
	info.w = W;
	info.h = H;
	info.n_frame = NFRAME;
	// fps = 24/1 = 24
	info.fps_n = 24;
	info.fps_d = 1;
};


void Lab2VideoGenerator::Generate(uint8_t *yuv) {
	//hipMemset(yuv, (impl->t)*255/NFRAME, W*H);
	
	
	//test
	
	
		
	

	int n = impl->p.size();
	//printf("length%d\n", sizeof(data));
	int *p_gpu;
	hipMalloc(&p_gpu, sizeof(int)*n);
	SyncedMemory<int> p_sync(impl->p.data(), p_gpu, n);
	p_sync.get_cpu_wo();
	//const int* temp = impl->s->get_cpu_ro();
	if(impl->t==0){
		for(int i=0;i<256;i++){			
			printf("%d ", p_sync.get_cpu_ro()[i]);
		}
	}
	//printf("output test\n");
	
	noise<<<W*H/64,64>>>((double)impl->t/NFRAME,p_sync.get_gpu_ro(),yuv);
	hipMemset(yuv+W*H, 91, W*H/4);
	hipMemset(yuv+W*H+W*H/4,167 , W*H/4);
	//printf("generate frame: %d\n", impl->t);

	++(impl->t);
}
