#include "hip/hip_runtime.h"
#include "lab3.h"
#include <cstdio>
#include <cmath>

__device__ __host__ int CeilDiv(int a, int b) { return (a-1)/b + 1; }
__device__ __host__ int CeilAlign(int a, int b) { return CeilDiv(a, b) * b; }

__global__ void SimpleClone(
	const float *background,
	const float *target,
	const float *mask,
	float *output,
	const int wb, const int hb, const int wt, const int ht,
	const int oy, const int ox
)
{
	const int yt = blockIdx.y * blockDim.y + threadIdx.y;
	const int xt = blockIdx.x * blockDim.x + threadIdx.x;
	const int curt = wt*yt+xt;
	if (yt < ht and xt < wt and mask[curt] > 127.0f) {
		const int yb = oy+yt, xb = ox+xt;
		const int curb = wb*yb+xb;
		if (0 <= yb and yb < hb and 0 <= xb and xb < wb) {
			output[curb*3+0] = target[curt*3+0];
			output[curb*3+1] = target[curt*3+1];
			output[curb*3+2] = target[curt*3+2];
		}
	}
}

__global__ void Upsample(
	float *target,
	float *up,
	const int wt, const int ht,
	const int wt_o,const int ht_o
	)
{
	const int yt = blockIdx.y * blockDim.y + threadIdx.y;
	const int xt = blockIdx.x * blockDim.x + threadIdx.x;
	//if(yt>=ht||xt>=wt)return;
	const int curt = wt*yt+xt;
	const int curu = 2*wt_o*yt+2*xt;
	if(curt>=wt*ht)return;

	int t1,t2,t3,t4;

	if(2*xt<wt_o && 2*yt<ht_o) {
		t1 = curu;
		t2 = curu+1;
		t3 = curu+wt_o;
		t4 = curu+wt_o+1;

	}
	else if(2*xt>=wt_o && 2*yt>=ht_o) {
		t1 = t2 = t3 = t4 = curu;
		//printf("error up %d\n",curt);

	}
	else if(2*xt>=wt_o) {
		t1 = curu;
		t2 = curu;
		t3 = curu+wt_o;
		t4 = curu+wt_o;
		//printf("error up %d\n",curt);


	}
	else if(2*yt>=ht_o) {
		t1 = curu;
		t2 = curu+1;
		t3 = curu;
		t4 = curu+1;
		//printf("error up %d\n",curt);

	}
	else{
		printf("error up %d\n",curt);
	}

	up[(3*t1)+0] = target[(3*curt)+0];up[(3*t2)+0] = target[(3*curt)+0];up[(3*t3)+0] = target[(3*curt)+0];up[(3*t4)+0] = target[(3*curt)+0];
	up[(3*t1)+1] = target[(3*curt)+1];up[(3*t2)+1] = target[(3*curt)+1];up[(3*t3)+1] = target[(3*curt)+1];up[(3*t4)+1] = target[(3*curt)+1];
	up[(3*t1)+2] = target[(3*curt)+2];up[(3*t2)+2] = target[(3*curt)+2];up[(3*t3)+2] = target[(3*curt)+2];up[(3*t4)+2] = target[(3*curt)+2];
	

}


__global__ void Downsample(
	const float *target,
	float *down,
	const int wt, const int ht,
	const int wt_o,const int ht_o
	)
{
	const int yt = blockIdx.y * blockDim.y + threadIdx.y;
	const int xt = blockIdx.x * blockDim.x + threadIdx.x;
	//if(yt>=ht||xt>=wt)return;
	const int curd = wt*yt+xt;
	const int curt = 2*wt_o*yt+2*xt;
	if(curd>=wt*ht)return;

	int t1,t2,t3,t4;

	if(2*xt<wt_o && 2*yt<ht_o) {
		t1 = curt;
		t2 = curt+1;
		t3 = curt+wt_o;
		t4 = curt+wt_o+1;

	}
	else if(2*xt>=wt_o && 2*yt>=ht_o) {
		t1 = t2 = t3 = t4 = curt;
		//printf("error down1 %d\n",curd);

	}
	else if(2*xt>=wt_o) {
		t1 = curt;
		t2 = curt;
		t3 = curt+wt_o;
		t4 = curt+wt_o;
		//printf("error down2 %d\n",curd);
	}
	else if(2*yt>=ht_o) {
		t1 = curt;
		t2 = curt+1;
		t3 = curt;
		t4 = curt+1;
		//printf("error down3 %d\n",curd);
	}
	else{
		printf("error down4 %d\n",curd);
	}

	down[(3*curd)+0] = (target[(3*t1)+0] + target[(3*t2)+0] + target[(3*t3)+0] + target[(3*t4)+0])/4;
	down[(3*curd)+1] = (target[(3*t1)+1] + target[(3*t2)+1] + target[(3*t3)+1] + target[(3*t4)+1])/4;
	down[(3*curd)+2] = (target[(3*t1)+2] + target[(3*t2)+2] + target[(3*t3)+2] + target[(3*t4)+2])/4;

}

__global__ void Downsample_mask(
	const float *target,
	float *down,
	const int wt, const int ht,
	const int wt_o,const int ht_o
	)
{
	const int yt = blockIdx.y * blockDim.y + threadIdx.y;
	const int xt = blockIdx.x * blockDim.x + threadIdx.x;
	const int curd = wt*yt+xt;
	const int curt = 2*wt_o*yt+2*xt;

	int t1,t2,t3,t4;

	if(2*xt<wt_o && 2*yt<ht_o) {
		t1 = curt;
		t2 = curt+1;
		t3 = curt+wt_o;
		t4 = curt+wt_o+1;

	}
	else if(2*xt>=wt_o && 2*yt>=ht_o) {
		t1 = t2 = t3 = t4 = curt;
	}
	else if(2*xt>=wt_o) {
		t1 = curt;
		t2 = curt;
		t3 = curt+wt_o;
		t4 = curt+wt_o;

	}
	else if(2*yt>=ht_o) {
		t1 = curt;
		t2 = curt+1;
		t3 = curt;
		t4 = curt+1;

	}
	else{
		printf("error\n");
	}

	down[curd] = target[t1] + target[t2] + target[t3] + target[t4];
	
	if(down[curd]>=255.0*2){
		down[curd]=255.0;
	}
	else{
		down[curd]=0.0;
	}
}

__global__ void merge(const float *target,float *output,const int wt,const int ht){
	const int yt = blockIdx.y * blockDim.y + threadIdx.y;
	const int xt = blockIdx.x * blockDim.x + threadIdx.x;
	const int curd = wt*yt+xt;
	if(abs(target[3*curd+0]+target[3*curd+1]+target[3*curd+2]-output[3*curd+0]+output[3*curd+1]+output[3*curd+2])<100.0){
	output[3*curd+0] = target[3*curd+0];
	output[3*curd+1] = target[3*curd+1];
	output[3*curd+2] = target[3*curd+2];
	//output[3*curd+0] = (output[3*curd+0]+target[3*curd+0])/2;
	//output[3*curd+1] = (output[3*curd+1]+target[3*curd+1])/2;
	//output[3*curd+2] = (output[3*curd+2]+target[3*curd+2])/2;
	}
	//else {
	//}
}

__global__	void CalculateFixed(
	const float *background,
	const float *target,
	const float *mask,
	float *fixed,
	const int wb, const int hb, const int wt, const int ht,
	const int oy, const int ox
)
{
	const int yt = blockIdx.y * blockDim.y + threadIdx.y;
	const int xt = blockIdx.x * blockDim.x + threadIdx.x;
	if(yt>=ht||xt>=wt)return;
	const int curt = wt*yt+xt;
	const int Tn = (yt-1<0)? curt:curt-wt, Ts = (ht<=yt+1)? curt:curt+wt, Tw = (xt-1<0)? curt:curt-1, Te = (wt<=xt+1)? curt:curt+1;

	fixed[curt*3+0] = 4*target[curt*3+0] - target[(Tn)*3+0] - target[(Ts)*3+0] - target[(Tw)*3+0] - target[(Te)*3+0];
	fixed[curt*3+1] = 4*target[curt*3+1] - target[(Tn)*3+1] - target[(Ts)*3+1] - target[(Tw)*3+1] - target[(Te)*3+1];
	fixed[curt*3+2] = 4*target[curt*3+2] - target[(Tn)*3+2] - target[(Ts)*3+2] - target[(Tw)*3+2] - target[(Te)*3+2];

	if(yt-1<0) {
		const int yb = (oy+yt-1<0)? oy+yt:oy+yt-1, xb = ox+xt;
		const int curb = wb*yb+xb;

		fixed[curt*3+0] += background[curb*3+0];
		fixed[curt*3+1] += background[curb*3+1];
		fixed[curt*3+2] += background[curb*3+2];
	}
	else if (mask[curt-wt] < 127.0f) {
		const int yb = oy+yt-1, xb = ox+xt;
		const int curb = wb*yb+xb;

		fixed[curt*3+0] += background[curb*3+0];
		fixed[curt*3+1] += background[curb*3+1];
		fixed[curt*3+2] += background[curb*3+2];
	}
	if(ht<=yt+1) {
		const int yb = (hb<=oy+yt+1)? oy+yt:oy+yt+1, xb = ox+xt;
		const int curb = wb*yb+xb;

		fixed[curt*3+0] += background[curb*3+0];
		fixed[curt*3+1] += background[curb*3+1];
		fixed[curt*3+2] += background[curb*3+2];
	}
	else if (mask[curt+wt] < 127.0f) {
		const int yb = oy+yt+1, xb = ox+xt;
		const int curb = wb*yb+xb;

		fixed[curt*3+0] += background[curb*3+0];
		fixed[curt*3+1] += background[curb*3+1];
		fixed[curt*3+2] += background[curb*3+2];
	}

	if(xt-1<0) {
		const int yb = oy+yt, xb = (ox+xt-1<0)? ox+xt:ox+xt-1;
		const int curb = wb*yb+xb;

		fixed[curt*3+0] += background[curb*3+0];
		fixed[curt*3+1] += background[curb*3+1];
		fixed[curt*3+2] += background[curb*3+2];

	}
	else if (mask[curt-1] < 127.0f) {
		const int yb = oy+yt, xb = ox+xt-1;
		const int curb = wb*yb+xb;
		fixed[curt*3+0] += background[curb*3+0];
		fixed[curt*3+1] += background[curb*3+1];
		fixed[curt*3+2] += background[curb*3+2];
	}
	if(wt<=xt+1) {
		const int yb = oy+yt, xb = (wb<=ox+xt+1)? ox+xt:ox+xt+1;
		const int curb = wb*yb+xb;

		fixed[curt*3+0] += background[curb*3+0];
		fixed[curt*3+1] += background[curb*3+1];
		fixed[curt*3+2] += background[curb*3+2];
	}
	else if (mask[curt+1] < 127.0f) {
		const int yb = oy+yt, xb = ox+xt+1;
		const int curb = wb*yb+xb;
		fixed[curt*3+0] += background[curb*3+0];
		fixed[curt*3+1] += background[curb*3+1];
		fixed[curt*3+2] += background[curb*3+2];
	}
}

__global__ void PoissonImageCloningIteration(
	float *fixed,
	const float *mask,
	float *buf1,
	float *buf2,
	const int wt,const int ht
	)
{
	const int yt = blockIdx.y * blockDim.y + threadIdx.y;
	const int xt = blockIdx.x * blockDim.x + threadIdx.x;
	if(yt>=ht||xt>=wt)return;
	const int curt = wt*yt+xt;
	buf2[curt*3+0] = fixed[curt*3+0];//+buf1[(curt-wt)*3+0]+buf1[(curt+wt)*3+0]+buf1[(curt-1)*3+0]+buf1[(curt+1)*3+0];
	buf2[curt*3+1] = fixed[curt*3+1];//+buf1[(curt-wt)*3+1]+buf1[(curt+wt)*3+1]+buf1[(curt-1)*3+1]+buf1[(curt+1)*3+1];
	buf2[curt*3+2] = fixed[curt*3+2];//+buf1[(curt-wt)*3+2]+buf1[(curt+wt)*3+2]+buf1[(curt-1)*3+2]+buf1[(curt+1)*3+2];

	if(yt-1>=0) {
		if (mask[curt-wt] > 127.0f) {
			const int cur = curt-wt; 
			buf2[curt*3+0] += (buf1[cur*3+0]);
			buf2[curt*3+1] += (buf1[cur*3+1]);
			buf2[curt*3+2] += (buf1[cur*3+2]);
		}
	}

	if(ht>yt+1) {	
		if (mask[curt+wt] > 127.0f) {
			const int cur = curt+wt;
			buf2[curt*3+0] += (buf1[cur*3+0]);
			buf2[curt*3+1] += (buf1[cur*3+1]);
			buf2[curt*3+2] += (buf1[cur*3+2]);
		}
	}

    if(xt-1>=0) {   
		if (mask[curt-1] > 127.0f) {
			const int cur = curt-1;
			buf2[curt*3+0] += (buf1[cur*3+0]);
			buf2[curt*3+1] += (buf1[cur*3+1]);
			buf2[curt*3+2] += (buf1[cur*3+2]);
		}
	}

	if(wt>xt+1) {
		if (mask[curt+1] > 127.0f) {
			const int cur = curt+1;
			buf2[curt*3+0] += (buf1[cur*3+0]);
			buf2[curt*3+1] += (buf1[cur*3+1]);
			buf2[curt*3+2] += (buf1[cur*3+2]);
		}
	}
	
	
	buf2[curt*3+0] /=4;
	buf2[curt*3+1] /=4;
	buf2[curt*3+2] /=4;

}

void PoissonImageCloning(
	const float *background,
	const float *target,
	const float *mask,
	float *output,
	const int wb, const int hb, const int wt, const int ht,
	const int oy, const int ox
)
{
	//hipMemcpy(output, background, wb*hb*sizeof(float)*3, hipMemcpyDeviceToDevice);
	//SimpleClone<<<dim3(CeilDiv(wt,32), CeilDiv(ht,16)), dim3(32,16)>>>(
	//	background, target, mask, output,
	//	wb, hb, wt, ht, oy, ox
	//);
	// set up
	float *fixed, *buf1, *buf2;
	hipMalloc(&fixed, 3*wt*ht*sizeof(float));
	hipMalloc(&buf1, 3*wt*ht*sizeof(float));
	hipMalloc(&buf2, 3*wt*ht*sizeof(float));

	//downsample

	//set downsample memory
	
	const int wt_d = CeilDiv(wt,2);
	const int ht_d = CeilDiv(ht,2);
	const int wb_d = CeilDiv(wb,2);
	const int hb_d = CeilDiv(hb,2);
	const int ox_d = CeilDiv(ox,2);
	const int oy_d = CeilDiv(oy,2);

	float *fixed_d,*buf1_d,*buf2_d,*mask_d, *bkg_d;
	hipMalloc(&fixed_d, 3*wt_d*ht_d*sizeof(float));
	hipMalloc(&buf1_d, 3*wt_d*ht_d*sizeof(float));
	hipMalloc(&buf2_d, 3*wt_d*ht_d*sizeof(float));
	hipMalloc(&mask_d, wt_d*ht_d*sizeof(float));
	hipMalloc(&bkg_d, 3*wb_d*hb_d*sizeof(float));
	//hipMalloc(&bkg_u,3*wb*hb*sizeof(float));

	const int wt_4 = CeilDiv(wt_d,2);const int wt_8 = CeilDiv(wt_4,2);
	const int ht_4 = CeilDiv(ht_d,2);const int ht_8 = CeilDiv(ht_4,2);
	const int wb_4 = CeilDiv(wb_d,2);const int wb_8 = CeilDiv(wb_4,2);
	const int hb_4 = CeilDiv(hb_d,2);const int hb_8 = CeilDiv(hb_4,2);
	const int ox_4 = CeilDiv(ox_d,2);const int ox_8 = CeilDiv(ox_4,2);
	const int oy_4 = CeilDiv(oy_d,2);const int oy_8 = CeilDiv(oy_4,2);

	float *fixed_4,*buf1_4,*buf2_4,*mask_4, *bkg_4;
	float *fixed_8,*buf1_8,*buf2_8,*mask_8, *bkg_8;

	hipMalloc(&fixed_4, 3*wt_4*ht_4*sizeof(float));
	hipMalloc(&buf1_4, 3*wt_4*ht_4*sizeof(float));
	hipMalloc(&buf2_4, 3*wt_4*ht_4*sizeof(float));
	hipMalloc(&mask_4, wt_4*ht_4*sizeof(float));
	hipMalloc(&bkg_4, 3*wb_4*hb_4*sizeof(float));
	hipMalloc(&fixed_8, 3*wt_8*ht_8*sizeof(float));
	hipMalloc(&buf1_8, 3*wt_8*ht_8*sizeof(float));
	hipMalloc(&buf2_8, 3*wt_8*ht_8*sizeof(float));
	hipMalloc(&mask_8, wt_8*ht_8*sizeof(float));
	hipMalloc(&bkg_8, 3*wb_8*hb_8*sizeof(float));
	
	/*
	fixed_4 = fixed_d + wt_d*ht_d; fixed_8 = fixed_4 + wt_4*ht_4;
	buf1_4  = buf1_d + wt_d*ht_d;  buf1_8  = buf1_4 + wt_4*ht_4;
	buf2_4  = buf2_d + wt_d*ht_d;  buf2_8  = buf2_4 + wt_4*ht_4;
	mask_4  = mask_d + wt_d*ht_d;  mask_8  = mask_4 + wt_4*ht_4;
	bkg_4  = bkg_d + wb_d*hb_d;    bkg_8  = bkg_4 + wb_d*hb_4;
	*/




	
	// initialize the iteration
	dim3 gdim(CeilDiv(wt,32), CeilDiv(ht,16)), bdim(32,16);

	dim3 gdim_d(CeilDiv(wt_d,32), CeilDiv(ht_d,16));
	dim3 gdim_bd(CeilDiv(wb_d,32),CeilDiv(hb_d,16));
	dim3 gdim_4(CeilDiv(wt_4,32), CeilDiv(ht_4,16));
	dim3 gdim_b4(CeilDiv(wb_4,32),CeilDiv(hb_4,16));
	dim3 gdim_8(CeilDiv(wt_8,32), CeilDiv(ht_8,16));
	dim3 gdim_b8(CeilDiv(wb_8,32),CeilDiv(hb_8,16));

	CalculateFixed<<<gdim, bdim>>>(
	background, target, mask, fixed,
	wb, hb, wt, ht, oy, ox
	);

	//X2
	//hipMemcpy(buf1, target, sizeof(float)*3*wt*ht, hipMemcpyDeviceToDevice);
	Downsample<<<gdim_bd, bdim>>>(background,bkg_d,wb_d,hb_d,wb,hb);
	Downsample<<<gdim_d, bdim>>>(target,buf1_d,wt_d,ht_d,wt,ht);
	Downsample_mask<<<gdim_d, bdim>>>(mask,mask_d,wt_d,ht_d,wt,ht);

	CalculateFixed<<<gdim_d, bdim>>>(
	bkg_d, buf1_d, mask_d, fixed_d,
	wb_d, hb_d, wt_d, ht_d, oy_d, ox_d
	);

	//X4
	
	Downsample<<<gdim_b4, bdim>>>(bkg_d,bkg_4,wb_4,hb_4,wb_d,hb_d);
	Downsample<<<gdim_4, bdim>>>(buf1_d,buf1_4,wt_4,ht_4,wt_d,ht_d);
	Downsample_mask<<<gdim_4, bdim>>>(mask_d,mask_4,wt_4,ht_4,wt_d,ht_d);
	
	CalculateFixed<<<gdim_d, bdim>>>(
	bkg_4, buf1_4, mask_4, fixed_4,
	wb_4, hb_4, wt_4, ht_4, oy_4, ox_4
	);
	
	//X8
	Downsample<<<gdim_b8, bdim>>>(bkg_4,bkg_8,wb_8,hb_8,wb_4,hb_4);
	Downsample<<<gdim_8, bdim>>>(buf1_4,buf1_8,wt_8,ht_8,wt_4,ht_4);
	Downsample_mask<<<gdim_8, bdim>>>(mask_4,mask_8,wt_8,ht_8,wt_4,ht_4);
	
	CalculateFixed<<<gdim_d, bdim>>>(
	bkg_8, buf1_8, mask_8, fixed_8,
	wb_8, hb_8, wt_8, ht_8, oy_8, ox_8
	);
	

	// downsample iterate

	for(int i = 0; i < 200; ++i ){
		PoissonImageCloningIteration<<<gdim_8, bdim>>>(
		fixed_8, mask_8, buf1_8, buf2_8, wt_8, ht_8
		);
		PoissonImageCloningIteration<<<gdim_4, bdim>>>(
		fixed_8, mask_8, buf2_8, buf1_8, wt_8, ht_8
		);
	}
	
	Upsample<<<gdim_8, bdim>>>(buf1_8,buf1_4,wt_8,ht_8,wt_4,ht_4);

	for(int i = 0; i < 20; ++i ){
		PoissonImageCloningIteration<<<gdim_4, bdim>>>(
		fixed_4, mask_4, buf1_4, buf2_4, wt_4, ht_4
		);
		PoissonImageCloningIteration<<<gdim_4, bdim>>>(
		fixed_4, mask_4, buf2_4, buf1_4, wt_4, ht_4
		);
	}
	
	Upsample<<<gdim_4, bdim>>>(buf1_4,buf1_d,wt_4,ht_4,wt_d,ht_d);

	
	for(int i = 0; i < 20; ++i ){
		PoissonImageCloningIteration<<<gdim_d, bdim>>>(
		fixed_d, mask_d, buf1_d, buf2_d, wt_d, ht_d
		);
		PoissonImageCloningIteration<<<gdim_d, bdim>>>(
		fixed_d, mask_d, buf2_d, buf1_d, wt_d, ht_d
		);
	}
	
	Upsample<<<gdim_d, bdim>>>(buf1_d,buf1,wt_d,ht_d,wt,ht);
	//merge<<<gdim, bdim>>>(target,buf1,wt,ht);
	//Upsample<<<gdim_bd, bdim>>>(bkg_d,bkg_u,wb_d,hb_d,wb,hb);

	
	
	// iterate
	
	for (int i = 0; i < 20; ++i) {
		PoissonImageCloningIteration<<<gdim, bdim>>>(
		fixed, mask, buf1, buf2, wt, ht
		);
		PoissonImageCloningIteration<<<gdim, bdim>>>(
		fixed, mask, buf2, buf1, wt, ht
		);
	}
	
	

	
	// copy the image back
	//Downsample<<<gdim_d, bdim>>>(fixed,buf1_d,wt_d,ht_d,wt,ht);
	//Upsample<<<gdim_d, bdim>>>(buf1_d,buf1,wt_d,ht_d,wt,ht);
	hipMemcpy(output, background, wb*hb*sizeof(float)*3, hipMemcpyDeviceToDevice);
	
	SimpleClone<<<gdim, bdim>>>(
	background, buf1, mask, output,
	wb, hb, wt, ht, oy, ox
	);
	
	
	// clean up
	hipFree(fixed);
	hipFree(buf1);
	hipFree(buf2);
	hipFree(buf1_d);
	hipFree(buf2_d);
	hipFree(fixed_d);
	hipFree(bkg_d);
	hipFree(mask_d);
	hipFree(buf1_4);
	hipFree(buf2_4);
	hipFree(fixed_4);
	hipFree(bkg_4);
	hipFree(mask_4);
	hipFree(buf1_8);
	hipFree(buf2_8);
	hipFree(fixed_8);
	hipFree(bkg_8);
	hipFree(mask_8);
	//hipFree(bkg_u);
}
