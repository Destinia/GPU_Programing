#include "hip/hip_runtime.h"
#include "counting.h"
#include <fstream>
#include <cstdio>
#include <cassert>
#include <thrust/scan.h>
#include <thrust/transform.h>
#include <thrust/functional.h>
#include <thrust/device_ptr.h>
#include <thrust/execution_policy.h>
#include <thrust/unique.h>
#include <thrust/copy.h>

using namespace std;
__device__ __host__ int CeilDiv(int a, int b) { return (a-1)/b + 1; }
__device__ __host__ int CeilAlign(int a, int b) { return CeilDiv(a, b) * b; }

struct is_one
  {
    __host__ __device__
    bool operator()(const int x)
    {
      return (x != 1);
    }
  };

 struct to_sym
 {
 	__host__ __device__
 	int operator()(const int x)
 	{
 		return -1;
 	}
 };
 struct is_not_sym
 {
 	__host__ __device__
 	bool operator()(const int x)
 	{
 		return (x != -1);
 	}
 };
 

__global__ void check_all(const char *text, int *pos, int text_size, int *segtree,int leaf_start){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(text[idx]=='\n'||idx>=text_size){
		segtree[idx+leaf_start] = 0;
	}
	else{
		segtree[idx+leaf_start] = 1;
	}

}

__global__ void buildTree(int *segtree,int leaf_start){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(segtree[(idx+leaf_start)*2+1] != 0 && segtree[(idx+leaf_start)*2+2] != 0){
		segtree[idx+leaf_start] = segtree[(idx+leaf_start)*2+1] + segtree[(idx+leaf_start)*2+2];
	}
	else{
		segtree[idx+leaf_start] = 0;
	}
}

__global__ void findlength(int *segtree,int *pos,int leaf_start,int leaf_end,int text_size){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(idx>=text_size)return;
	int pivot = idx + leaf_start;
	int next;
	pos[idx] = 0;
	//bottom-up
	while(pivot != 0){
		//right child
		if(pivot%2 == 0){
			if(segtree[pivot]==0){
				break;
			}
			else{
				next = (pivot-1)/2;
				if(segtree[next]==0){
					pos[idx] += segtree[pivot];
					next = next*2+1;
				}
			}
		}
		//left child
		else{
			if(segtree[pivot]==0){
				break;
			}
			else{
				next = (pivot-1)/2 -1;
				pos[idx] += segtree[pivot];
			}

		}
	pivot = next;
	}
	//top-down
	while(pivot<=leaf_end){
		if(segtree[pivot] == 0){
			//find right child
			next = (pivot+1)*2;
		}
		else{
			pos[idx] += segtree[pivot];
			next = pivot*2;
		}
		pivot = next;
	}
}

__global__ void switch_char(char *ref,int *pos, char *out, int text_size){
	int idx = (blockIdx.x * blockDim.x + threadIdx.x)*2;
	if(pos[idx]==0||idx>=text_size){
		return;
	}
	if(pos[idx] % 2== 1){
		if(pos[idx+1]!=0){
			out[idx] = ref[idx+1];
			out[idx+1] = ref[idx] ;
		}
	}
	else {
		out[idx-1] = ref[idx];
		out[idx] = ref[idx-1];
	}
}

void CountPosition(const char *text, int *pos, int text_size)
{	
	int level = 0;
	int temp = text_size-1;
	int leaf_start=0;
	while(1){
		temp/=2;
		level++;
		if(temp==0)break;
	}
	int pivot = 1;
	for(int i=0;i<level;i++){
		leaf_start +=pivot;
		pivot*=2;
	}
	int length = leaf_start + pivot;
	int start  = leaf_start;
	//printf("\nlength %d\nlevel %d\npivot %d\nstart %d\n",length ,level,pivot,start);
	//create segtree memory
	int *seg_tree;
	hipMalloc((void**)&seg_tree,length*sizeof(int));

	
	check_all<<<(pivot>>6),64>>>(text,pos,text_size,seg_tree,start);

	hipDeviceSynchronize();
	for(int i=0;i<level;i++){
		pivot/=2;
		leaf_start -=pivot;
		if(pivot>=64)
			buildTree<<<(pivot>>6),64>>>(seg_tree,leaf_start);
		else
			buildTree<<<1,pivot>>>(seg_tree,leaf_start);
	}
	//for debug
	/*
	int check = 1;
	int count = 0;
	int *check_tree = new int[length];
	hipMemcpy(check_tree,seg_tree,length*sizeof(int),hipMemcpyDeviceToHost);
	fstream fp;
	fp.open("tree.txt",ios::out);
	for(int i=0;i<length;i++){
		if(count == check ){check*=2;count=0;fp<<endl;}
		fp<<check_tree[i]<<" ";
		count++;
	}
	fp.close();
	delete[] check_tree;
	*/
	////
	
	hipDeviceSynchronize();
	
	findlength<<<(text_size/64+1),64>>>(seg_tree,pos,start,length,text_size);

	hipFree(seg_tree);
}

int ExtractHead(const int *pos, int *head, int text_size)
{
	int *buffer;
	int nhead;
	hipMalloc(&buffer, sizeof(int)*text_size*2); // this is enough
	thrust::device_ptr<const int> pos_d(pos);
	thrust::device_ptr<int> head_d(head), flag_d(buffer), cumsum_d(buffer+text_size);
	thrust::device_ptr<int> temp;
	thrust::sequence(flag_d,flag_d+text_size);
 

  	thrust::transform_if(flag_d, flag_d + text_size, pos_d, flag_d, to_sym(), is_one());
  	temp = thrust::copy_if(flag_d,flag_d+text_size,head_d,is_not_sym());
  	nhead = temp - head_d ;


	// TODO

	hipFree(buffer);
	return nhead;
}

void Part3(char *text, int *pos, int *head, int text_size, int n_head)
{
	char *temp;
	hipMalloc(&temp,sizeof(char)*text_size);
	hipMemcpy(temp,text,text_size*sizeof(char),hipMemcpyDeviceToDevice);
	switch_char<<<text_size/64+1,64>>>(temp,pos,text,text_size);

	//check
	/*char *check = new char[text_size];
	hipMemcpy(check,text,text_size*sizeof(char),hipMemcpyDeviceToHost);
	fstream fp;
	fp.open("switch.txt",ios::out);
	for(int i=0;i<text_size;i++){
		fp<<check[i]<<' ';
	}
	fp.close();*/

	hipFree(temp);
}
